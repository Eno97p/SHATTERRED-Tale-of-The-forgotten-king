#include "hip/hip_runtime.h"
#include<Windows.h>
#include"TestCuda.cuh"





__constant__ float3 g_cameraPos;
__constant__ float g_maxDistanceSquared;



__device__ int atomicAdd(int* address, int val);

__global__ void cullingKernel(cu_VTXMATRIX* instances, int numInstances, float3 cameraPos, float maxDistanceSquared, int* visibleCount)
{

	unsigned int index_X = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (index_X < numInstances)
	{
		float4 instancePos = make_float4(instances[index_X].vTranslation.x, instances[index_X].vTranslation.y, instances[index_X].vTranslation.z, instances[index_X].vTranslation.w);
		float distanceSquared = (instancePos.x - cameraPos.x) * (instancePos.x - cameraPos.x) + 
								(instancePos.y - cameraPos.y) * (instancePos.y - cameraPos.y) +
								(instancePos.z - cameraPos.z) * (instancePos.z - cameraPos.z);

		if (distanceSquared < maxDistanceSquared)
		{
			int visibleIndex = atomicAdd(visibleCount, 1);
			if (visibleIndex != index_X)
			{
				// Swap
				cu_VTXMATRIX temp = instances[visibleIndex];
				instances[visibleIndex] = instances[index_X];
				instances[index_X] = temp;
			}
		}

	}
}

hipError_t LaunchCullingKernel(cu_VTXMATRIX* d_instanceData, int numInstances, float3 cameraPos, float maxDistanceSquared, int* d_visibleCount)
{
	const DWORD MAX_NUM_PER_ONCE = 65536 * 10;
	const DWORD THREAD_NUM_PER_BLOCK = 1024;
	hipError_t cudaStatus;

	while (numInstances)
	{
		DWORD NumPerOnce = numInstances;
		if(NumPerOnce > MAX_NUM_PER_ONCE)
		{
			NumPerOnce = MAX_NUM_PER_ONCE;
		}

		dim3 threadPerBlock(1, 1);
		dim3 blockPerGrid(1, 1, 1);

		threadPerBlock.x = THREAD_NUM_PER_BLOCK;
		threadPerBlock.y = 1;

		blockPerGrid.x = (NumPerOnce / THREAD_NUM_PER_BLOCK) + ((NumPerOnce % THREAD_NUM_PER_BLOCK) != 0);
		blockPerGrid.y = 1;

		cullingKernel<<< blockPerGrid , threadPerBlock>>> (d_instanceData, NumPerOnce, cameraPos, maxDistanceSquared, d_visibleCount);
		cudaStatus = hipDeviceSynchronize();
		if(cudaStatus != hipSuccess)
		{
			return cudaStatus;
		}
		d_instanceData += NumPerOnce;
		numInstances -= NumPerOnce;
	}
	
	return cudaStatus;

}

